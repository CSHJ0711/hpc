
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 10000
#define Threads 100

__global__ void saxpy(float *a,float *b,float x,float y)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < N)
   {
      b[i] = a[i] * x + y;
   }
}
int main()
{
   float A[N],B[N],B2[N],X,Y;
   int i;
   float *A_d, *B_d;

   hipSetDevice(0);
   
   for ( i = 0; i < N; i++)
      A[i] = i * 2;

   X = 1.23;
   Y = 2.34;
   
   hipMalloc((void **)&A_d, N * sizeof(float));
   hipMalloc((void **)&B_d, N * sizeof(float));

   hipMemcpy(A_d, A, N * sizeof(float),hipMemcpyHostToDevice);

   dim3 dimBlock(Threads,1,1);
   dim3 dimGrid((N + Threads - 1) / Threads,1,1);

   saxpy<<<dimGrid, dimBlock>>>(A_d,B_d,X,Y);

   hipMemcpy(B, B_d, N * sizeof(float),hipMemcpyDeviceToHost);

   hipFree(A_d);
   hipFree(B_d);

   for ( i = 0; i < N; i++)
   {
      B2[i] = A[i]*X + Y; //B2 is used for checking
   }

   //check the results
   for ( i = 0; i < N; i++)
      if (fabs(B[i] - B2[i]) > 0.01 )
         printf("%d: %f %f\n",i,B[i],B2[i]);

}
