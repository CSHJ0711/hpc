
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1280000
#define THREADS 128

__device__ float f(float x)
{
   double temp;
   temp = x*x+2;
   return x/(temp*temp*temp);
}

__global__ void area_kernel(float *sums)
{
   float a, b, dx, x;
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   int j;
   __shared__ float sdata[THREADS];

   //2) complelte
   a = 0.0;
   b = 2.0;
   dx = (b - a) / N;

   x = a + i * dx;
   sdata[threadIdx.x] = 0.5 * (f(x) + f(x+dx)) * dx;;
   __syncthreads();

   for (j = blockDim.x / 2; j > 0; j >>= 1)
   {
      if (threadIdx.x < j)
      {
         sdata[threadIdx.x] += sdata[threadIdx.x + j];
      }
      __syncthreads();
   }
   if(threadIdx.x == 0)
   {
      sums[blockIdx.x] = sdata[0];
   }

}

int main()
{
   float *sums, *sums_d, area, elapsed;
   int i;

   hipSetDevice(0);

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N + dimBlock.x-1)/dimBlock.x);

   printf("dimBlock.x :: %d\n",dimBlock.x);
   printf("dimBlock.y :: %d\n",dimBlock.y);
   printf("dimGrid :: %d\n",dimGrid.y);

   hipEvent_t start, stop;

   hipMalloc((void **)&sums_d, sizeof(float)*dimGrid.x);
   sums = (float *)malloc(sizeof(float)*dimGrid.x);

   //3) COMPLETE (timeing - start)
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start,0);

   //4) Complete (call GPU function)
   area_kernel<<<dimGrid, dimBlock>>>(sums_d);

   //copy values from GPU memovry to CPU memory
   hipMemcpy(sums,sums_d,sizeof(float)*dimGrid.x,hipMemcpyDeviceToHost);

   //5) Complete (timeing - stop)
   hipEventRecord(stop,0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed,start,stop);

   //add the computed value to the value of pi
   area = 0.0;
   for ( i = 0 ; i < dimGrid.x; i++)
      area += sums[i];

   printf("area : %5.10f\n",area);
   printf("elpased time: %f milliseconds\n",elapsed);
   printf("GFLOPS : %5.2f\n", (N*16.0/(elapsed/1000.0))/1000000000.0);

   hipFree(sums_d);
   free(sums);
}
