#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define N 1280000
#define THREADS 128

__device__ float f(float x)
{
   //1) complelte
}

__global__ void area_kernel(float *sums)
{
   float a, b, dx, x, y;
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   __shared__ float sdata[THREADS];

   //2) complelte
   sdata[threadIdx.x] = ...;

}

int main()
{
   float *sums, *sums_d, area, elapsed;
   int i;

   hipSetDevice(0);

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N + dimBlock.x-1)/dimBlock.x);


   hipEvent_t start, stop;
   hipMalloc((void **)&sums_d, sizeof(float)*dimGrid.x);
   sums = (float *)malloc(sizeof(float)*dimGrid.x);

   //3) COMPLETE (timeing - start)

   //4) Complete (call GPU function)
   area_kernel<<<dimGrid, dimBlock>>>(sums_d);

   //copy values from GPU memovry to CPU memory
   hipMemcpy(sums,sums_d,sizeof(float)*dimGrid.x,hipMemcpyDeviceToHost);

  //5) Complete (timeing - stop)

   //add the computed value to the value of pi
   area = 0.0;
   for ( i = 0 ; i < dimGrid.x; i++)
      area += sums[i];

   printf("area : %5.10f\n",area);
   printf("elpased time: %f milliseconds\n",elapsed);
   printf("GFLOPS : %5.2f\n", (N*16.0/(elapsed/1000.0))/1000000000.0);

   hipFree(sums_d);
   free(sums);
}
