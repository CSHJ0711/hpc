
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS 10
#define N 10000

__global__ void vecAdd(float *A, float *B, float *C)
{
   //CUDA kernel
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   if ( i < N)
      C[i] = A[i] + B[i];
}

int main()
{
   float A[N], B[N], C[N], C2[N], *A_d, *B_d, *C_d;
   int i;

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N + dimBlock.x-1)/dimBlock.x);
   hipSetDevice(0);

   for ( i = 0; i < N; i++)
   {
      A[i] = i * 2;
      B[i] = N - i;
      C2[i] = A[i] + B[i];; //C2 is used to check the results
   }
   //alocate array on device
   hipMalloc((void **)&A_d, N * sizeof(float)); 
   hipMalloc((void **)&B_d, N * sizeof(float)); 
   hipMalloc((void **)&C_d, N * sizeof(float)); 
   //copy array from host memory to device memory
   hipMemcpy(A_d, A, N * sizeof(float), hipMemcpyHostToDevice); 
   hipMemcpy(B_d, B, N * sizeof(float), hipMemcpyHostToDevice); 
   //call GPU function(vecAdd)
   vecAdd<<<dimGrid, dimBlock>>>(A_d, B_d, C_d); 
   //Copy the result back
   hipMemcpy(C, C_d, N * sizeof(float), hipMemcpyDeviceToHost); 
   //Check the results
   for ( i = 0; i < N; i++)
      if (C[i] != C2[i])
         printf("%d %f %f\n",i,C[i],C2[i]);

   //free memory on the device
   hipFree(A_d);
   hipFree(B_d);
   hipFree(C_d);

   return 0;
}
